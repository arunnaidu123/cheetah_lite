#include "hip/hip_runtime.h"
/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2016 The SKA organisation
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include "cheetah/Configuration.h"

#ifdef SKA_CHEETAH_ENABLE_CUDA
#ifdef __HIPCC__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace ska {
namespace cheetah {
namespace corner_turn {
namespace cuda {

template <typename InputNumericalT, typename OutputNumericalT>
__global__ void simple_corner_turn_kernel(const InputNumericalT *d_input, OutputNumericalT *d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if(t < first_dimension*second_dimension)
    {
        int c = t%first_dimension;
        int s = t/first_dimension;
        d_output[c * second_dimension + s] = (OutputNumericalT)(d_input[s * first_dimension + c]);
    }
}

} //namespace cuda
} //namespace corner_turn
} //namespace cheetah
} //namespace ska

#endif // __HIPCC__
#endif // SKA_CHEETAH_ENABLE_CUDA
