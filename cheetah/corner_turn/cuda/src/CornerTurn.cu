/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2016 The SKA organisation
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include "cheetah/Configuration.h"
#ifdef SKA_CHEETAH_ENABLE_CUDA

#include "cheetah/corner_turn/cuda/detail/corner_turn_kernels.cu"
#include "cheetah/corner_turn/cuda/CornerTurn.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace ska {
namespace cheetah {
namespace corner_turn {
namespace cuda {

template <typename InputNumericalT, typename OutputNumericalT>
void corner_turn_impl(const InputNumericalT* d_input
                     , OutputNumericalT* d_output
                     , std::size_t first_dimension
                     , std::size_t second_dimension)
{
    int max_threads_per_block;
    int device;
    hipGetDevice(&device);
    hipDeviceGetAttribute(&max_threads_per_block,
    hipDeviceAttributeMaxThreadsPerBlock, device);
    int num_blocks = (first_dimension*second_dimension)/max_threads_per_block;
    if((first_dimension*second_dimension)%max_threads_per_block != 0) ++num_blocks;
    simple_corner_turn_kernel<<<num_blocks, max_threads_per_block>>>(d_input, d_output, first_dimension, second_dimension);
    hipDeviceSynchronize();
}


void cuda_corner_turn(const uint8_t* d_input, uint8_t* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

void cuda_corner_turn(const uint8_t* d_input, uint16_t* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

void cuda_corner_turn(const uint8_t* d_input, float* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

void cuda_corner_turn(const uint16_t* d_input, uint8_t* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

void cuda_corner_turn(const uint16_t* d_input, uint16_t* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

void cuda_corner_turn(const uint16_t* d_input, float* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

void cuda_corner_turn(const float* d_input, uint8_t* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

void cuda_corner_turn(const float* d_input, uint16_t* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

void cuda_corner_turn(const float* d_input, float* d_output, std::size_t first_dimension, std::size_t second_dimension)
{
    corner_turn_impl(d_input, d_output, first_dimension, second_dimension);
}

} //namespace cuda
} //namespace corner_turn
} //namespace cheetah
} //namespace ska

#endif // SKA_CHEETAH_ENABLE_CUDA
