
#include "cheetah/data/cuda/Series.h"

#ifdef SKA_CHEETAH_ENABLE_CUDA

#include "cheetah/cuda_utils/cuda_thrust.h"
#include <panda/arch/nvidia/DeviceCopy.h>

namespace ska {
namespace cheetah {
namespace data {

#ifdef __HIPCC__
#if __CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2)
#define __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
#endif

template <typename ValueType, typename Alloc>
Series<cheetah::Cuda, ValueType, Alloc>::Series(std::size_t size)
    : BaseT(size)
{
}

template <typename ValueType, typename Alloc>
Series<cheetah::Cuda, ValueType, Alloc>::Series(std::size_t size, Allocator const& allocator)
#ifdef __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
    : BaseT(size, allocator)
{
#else // __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
    : BaseT(size)
{
    (void) allocator;
#endif // __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
}

template <typename ValueType, typename Alloc>
Series<cheetah::Cuda, ValueType, Alloc>::Series(Allocator const& allocator)
#ifdef __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
    : BaseT(allocator)
{
#else // __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
{
    (void) allocator;
#endif // __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
}

template<typename ValueType, typename Alloc>
template<typename OtherArch, typename OtherValueT, typename OtherAlloc>
Series<cheetah::Cuda, ValueType, Alloc>::Series(Series<OtherArch, OtherValueT, OtherAlloc> const& copy, Alloc const& allocator)
#ifdef __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
    : BaseT(copy.size(), allocator)
{
#else // __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
    : BaseT(copy.size())
{
    (void) allocator;
#endif // __NVCC_THRUST_SUPPORTS_ALLOCATOR_CONSTRUCTORS__
    panda::copy(copy.begin(), copy.end(), this->begin());
}

template <typename ValueType, typename Alloc>
Series<cheetah::Cuda, ValueType, Alloc>::~Series()
{
}

template<typename ValueType, typename Alloc>
template<typename OtherArch, typename OtherValueT, typename OtherAlloc>
Series<cheetah::Cuda, ValueType, Alloc>& Series<cheetah::Cuda, ValueType, Alloc>::operator=(Series<OtherArch, OtherValueT, OtherAlloc> const& copy)
{
    this->resize(copy.size());
    panda::copy(copy.begin(), copy.end(), this->begin());
    return *this;
}

#endif // __HIPCC__

} // namespace data
} // namespace cheetah
} // namespace ska

#endif //SKA_CHEETAH_ENABLE_CUDA
